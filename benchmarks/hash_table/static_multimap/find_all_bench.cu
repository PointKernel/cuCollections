/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <nvbench/nvbench.cuh>

#include <thrust/device_vector.h>
#include <random>

#include "cuco/static_multimap.cuh"

/**
 * @brief Generates input keys by a given number of repetitions per key.
 *
 */
template <typename Key, typename OutputIt>
static void generate_multikeys(OutputIt output_begin, OutputIt output_end, size_t const num_reps)
{
  auto num_keys = std::distance(output_begin, output_end);

  for (auto i = 0; i < num_keys; ++i) {
    output_begin[i] = (i % (num_keys / num_reps)) + 1;
  }
}

/**
 * @brief A benchmark evaluating multi-value retrieval performance by varing number of repetitions
 * per key:
 * - 100'000'000 keys are inserted
 * - Map occupancy is fixed at 0.4
 * - Number of repetitions per key: 1, ... , 128, 256
 *
 */
template <typename Key, typename Value, nvbench::int32_t CGSize, nvbench::int32_t BufferSize>
std::enable_if_t<(sizeof(Key) == sizeof(Value)), void> nvbench_find_all(
  nvbench::state& state,
  nvbench::type_list<Key, Value, nvbench::enum_type<CGSize>, nvbench::enum_type<BufferSize>>)
{
  std::size_t const num_keys = state.get_int64("NumInputs");
  auto const occupancy       = state.get_float64("Occupancy");
  std::size_t const size     = num_keys / occupancy;
  std::size_t const num_reps = state.get_int64("NumReps");

  constexpr bool is_outer = true;

  std::vector<Key> h_keys(num_keys);
  std::vector<cuco::pair_type<Key, Value>> h_pairs(num_keys);

  generate_multikeys<Key>(h_keys.begin(), h_keys.end(), num_reps);

  for (auto i = 0; i < num_keys; ++i) {
    Key key           = h_keys[i];
    Value val         = h_keys[i];
    h_pairs[i].first  = key;
    h_pairs[i].second = val;
  }

  // Get an array of unique keys
  std::set<Key> key_set(h_keys.begin(), h_keys.end());
  std::vector<Key> h_unique_keys(key_set.begin(), key_set.end());
  thrust::device_vector<Key> d_unique_keys(h_unique_keys);

  thrust::device_vector<cuco::pair_type<Key, Value>> d_results(2 * num_keys);
  thrust::device_vector<cuco::pair_type<Key, Value>> d_pairs(h_pairs);

  state.add_element_count(num_keys, "NumKeys");
  state.add_global_memory_writes<Key>(num_keys * 2);

  state.exec(
    nvbench::exec_tag::sync | nvbench::exec_tag::timer, [&](nvbench::launch& launch, auto& timer) {
      cuco::static_multimap<Key, Value, cuco::double_hashing<Key, Value, CGSize>> map{size, -1, -1};
      map.insert(d_pairs.begin(), d_pairs.end());

      timer.start();
      auto view = map.get_device_view();

      auto const block_size  = 128;
      auto const warp_size   = 32;
      auto const buffer_size = CGSize * BufferSize;
      auto const stride      = 1;
      auto const grid_size = (CGSize * num_keys + stride * block_size - 1) / (stride * block_size);

      using KeyEqual = thrust::equal_to<Key>;

      KeyEqual key_equal;

      using atomic_ctr_type = typename cuco::static_multimap<Key, Value>::atomic_ctr_type;
      atomic_ctr_type* num_items;
      CUCO_CUDA_TRY(hipMallocManaged(&num_items, sizeof(atomic_ctr_type)));
      *num_items = 0;
      int device_id;
      CUCO_CUDA_TRY(hipGetDevice(&device_id));
      CUCO_CUDA_TRY(hipMemPrefetchAsync(num_items, sizeof(atomic_ctr_type), device_id));

      // Use timers to explicitly mark the target region
      cuco::detail::vectorized_retrieve<block_size, warp_size, CGSize, buffer_size, is_outer>
        <<<grid_size, block_size, 0, launch.get_stream()>>>(d_unique_keys.begin(),
                                                            d_unique_keys.end(),
                                                            d_results.data().get(),
                                                            num_items,
                                                            view,
                                                            key_equal);
      CUCO_CUDA_TRY(hipDeviceSynchronize());
      timer.stop();

      CUCO_CUDA_TRY(hipFree(num_items));
    });
}

template <typename Key, typename Value, nvbench::int32_t CGSize, nvbench::int32_t BufferSize>
std::enable_if_t<(sizeof(Key) != sizeof(Value)), void> nvbench_find_all(
  nvbench::state& state,
  nvbench::type_list<Key, Value, nvbench::enum_type<CGSize>, nvbench::enum_type<BufferSize>>)
{
  state.skip("Key should be the same type as Value.");
}

using key_type    = nvbench::type_list<nvbench::int32_t, nvbench::int64_t>;
using value_type  = nvbench::type_list<nvbench::int32_t, nvbench::int64_t>;
using cg_size     = nvbench::enum_type_list<1, 2, 4, 8, 16, 32>;
using buffer_size = nvbench::enum_type_list<1, 2, 4, 8, 16>;

NVBENCH_BENCH_TYPES(nvbench_find_all,
                    NVBENCH_TYPE_AXES(key_type, value_type, cg_size, nvbench::enum_type_list<2>))
  .set_type_axes_names({"Key", "Value", "CGSize", "BufferSize"})
  .set_timeout(100)                            // Custom timeout: 100 s. Default is 15 s.
  .set_max_noise(3)                            // Custom noise: 3%. By default: 0.5%.
  .add_int64_axis("NumInputs", {100'000'000})  // Total number of key/value pairs: 100'000'000
  .add_float64_axis("Occupancy", {0.4})
  .add_int64_power_of_two_axis("NumReps", nvbench::range(0, 8, 1));

NVBENCH_BENCH_TYPES(
  nvbench_find_all,
  NVBENCH_TYPE_AXES(key_type, value_type, nvbench::enum_type_list<8>, buffer_size))
  .set_type_axes_names({"Key", "Value", "CGSize", "BufferSize"})
  .set_timeout(100)                            // Custom timeout: 100 s. Default is 15 s.
  .set_max_noise(3)                            // Custom noise: 3%. By default: 0.5%.
  .add_int64_axis("NumInputs", {100'000'000})  // Total number of key/value pairs: 100'000'000
  .add_float64_axis("Occupancy", {0.4})
  .add_int64_power_of_two_axis("NumReps", nvbench::range(0, 8, 1));
