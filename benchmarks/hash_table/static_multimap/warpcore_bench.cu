/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <random>

#include <thrust/device_vector.h>
#include <nvbench/nvbench.cuh>

#include <key_generator.hpp>
#include <warpcore.cuh>

using namespace warpcore;

NVBENCH_DECLARE_ENUM_TYPE_STRINGS(
  // Enum type:
  dist_type,
  // Callable to generate input strings:
  // Short identifier used for tables, command-line args, etc.
  // Used when context is available to figure out the enum type.
  [](dist_type d) {
    switch (d) {
      case dist_type::GAUSSIAN: return "GAUSSIAN";
      case dist_type::GEOMETRIC: return "GEOMETRIC";
      case dist_type::UNIFORM: return "UNIFORM";
      default: return "ERROR";
    }
  },
  // Callable to generate descriptions:
  // If non-empty, these are used in `--list` to describe values.
  // Used when context may not be available to figure out the type from the
  // input string.
  // Just use `[](auto) { return std::string{}; }` if you don't want these.
  [](auto) { return std::string{}; })

/**
 * @brief A benchmark evaluating multi-value `insert` performance:
 * - Total number of insertions: 100'000'000
 * - CG size: 8
 */
template <typename Key, typename Value, dist_type Dist, nvbench::int32_t Multiplicity>
std::enable_if_t<(sizeof(Key) == sizeof(Value)), void> nvbench_warpcore_insert(
  nvbench::state& state,
  nvbench::type_list<Key, Value, nvbench::enum_type<Dist>, nvbench::enum_type<Multiplicity>>)
{
  using hash_table_t = MultiValueHashTable<Key,
                                           Value,
                                           defaults::empty_key<Key>(),
                                           defaults::tombstone_key<Key>(),
                                           defaults::probing_scheme_t<Key, 8>,
                                           storage::key_value::AoSStore<Key, Value>>;

  auto const num_keys  = state.get_int64("NumInputs");
  auto const occupancy = state.get_float64("Occupancy");

  std::size_t const capacity = num_keys / occupancy;

  std::vector<Key> h_keys(num_keys);

  generate_keys<Dist, Multiplicity, Key>(h_keys.begin(), h_keys.end());

  thrust::device_vector<Key> d_keys(h_keys);
  thrust::device_vector<Key> d_values(h_keys);

  state.add_element_count(num_keys, "NumKeys");

  hash_table_t hash_table(capacity);

  state.exec(
    nvbench::exec_tag::sync | nvbench::exec_tag::timer, [&](nvbench::launch& launch, auto& timer) {
      hash_table.init();

      // Use timers to explicitly mark the target region
      timer.start();
      hash_table.insert(d_keys.data().get(), d_values.data().get(), num_keys, launch.get_stream());
      timer.stop();

      auto status = hash_table.pop_status(launch.get_stream());
      if (status.has_any_errors()) { std::cout << status << "\n"; }
    });
}

template <typename Key, typename Value, dist_type Dist, nvbench::int32_t Multiplicity>
std::enable_if_t<(sizeof(Key) != sizeof(Value)), void> nvbench_warpcore_insert(
  nvbench::state& state,
  nvbench::type_list<Key, Value, nvbench::enum_type<Dist>, nvbench::enum_type<Multiplicity>>)
{
  state.skip("Key should be the same type as Value.");
}

/**
 * @brief A benchmark evaluating multi-value `count` performance:
 * - Total number of insertions: 100'000'000
 * - CG size: 8
 */
template <typename Key, typename Value, dist_type Dist, nvbench::int32_t Multiplicity>
std::enable_if_t<(sizeof(Key) == sizeof(Value)), void> nvbench_warpcore_count(
  nvbench::state& state,
  nvbench::type_list<Key, Value, nvbench::enum_type<Dist>, nvbench::enum_type<Multiplicity>>)
{
  using hash_table_t = MultiValueHashTable<Key,
                                           Value,
                                           defaults::empty_key<Key>(),
                                           defaults::tombstone_key<Key>(),
                                           defaults::probing_scheme_t<Key, 8>,
                                           storage::key_value::AoSStore<Key, Value>>;

  auto const num_keys      = state.get_int64("NumInputs");
  auto const occupancy     = state.get_float64("Occupancy");
  auto const matching_rate = state.get_float64("MatchingRate");

  std::size_t const capacity = num_keys / occupancy;

  std::vector<Key> h_keys(num_keys);

  generate_keys<Dist, Multiplicity, Key>(h_keys.begin(), h_keys.end());

  thrust::device_vector<Key> d_keys(h_keys);
  thrust::device_vector<Key> d_values(h_keys);

  hash_table_t hash_table(capacity);

  hash_table.insert(d_keys.data().get(), d_values.data().get(), num_keys);

  generate_prob_keys<Key>(matching_rate, h_keys.begin(), h_keys.end());

  thrust::device_vector<Key> d_prob_keys(h_keys);
  thrust::device_vector<std::size_t> d_offsets(num_keys);

  state.add_element_count(num_keys, "NumKeys");

  state.exec(nvbench::exec_tag::sync | nvbench::exec_tag::timer,
             [&](nvbench::launch& launch, auto& timer) {
               std::size_t value_size_out = 0;

               timer.start();
               // nullptr to launch the dry-run count
               hash_table.retrieve(d_prob_keys.data().get(),
                                   num_keys,
                                   d_offsets.data().get(),
                                   d_offsets.data().get() + 1,
                                   nullptr,
                                   value_size_out,
                                   launch.get_stream());
               timer.stop();

               auto status = hash_table.pop_status(launch.get_stream());
               if (status.has_any_errors()) { std::cout << status << "\n"; }
             });
}

template <typename Key, typename Value, dist_type Dist, nvbench::int32_t Multiplicity>
std::enable_if_t<(sizeof(Key) != sizeof(Value)), void> nvbench_warpcore_count(
  nvbench::state& state,
  nvbench::type_list<Key, Value, nvbench::enum_type<Dist>, nvbench::enum_type<Multiplicity>>)
{
  state.skip("Key should be the same type as Value.");
}

/**
 * @brief A benchmark evaluating multi-value `find_all` performance:
 * - Total number of insertions: 100'000'000
 * - CG size: 8
 */
template <typename Key, typename Value, dist_type Dist, nvbench::int32_t Multiplicity>
std::enable_if_t<(sizeof(Key) == sizeof(Value)), void> nvbench_warpcore_retrieve(
  nvbench::state& state,
  nvbench::type_list<Key, Value, nvbench::enum_type<Dist>, nvbench::enum_type<Multiplicity>>)
{
  using hash_table_t = MultiValueHashTable<Key,
                                           Value,
                                           defaults::empty_key<Key>(),
                                           defaults::tombstone_key<Key>(),
                                           defaults::probing_scheme_t<Key, 8>,
                                           storage::key_value::AoSStore<Key, Value>>;

  auto const num_keys      = state.get_int64("NumInputs");
  auto const occupancy     = state.get_float64("Occupancy");
  auto const matching_rate = state.get_float64("MatchingRate");

  std::size_t const capacity = num_keys / occupancy;

  std::vector<Key> h_keys(num_keys);

  generate_keys<Dist, Multiplicity, Key>(h_keys.begin(), h_keys.end());

  thrust::device_vector<Key> d_keys(h_keys);
  thrust::device_vector<Key> d_values(h_keys);

  hash_table_t hash_table(capacity);

  hash_table.insert(d_keys.data().get(), d_values.data().get(), num_keys);

  generate_prob_keys<Key>(matching_rate, h_keys.begin(), h_keys.end());

  thrust::device_vector<Key> d_prob_keys(h_keys);

  state.add_element_count(num_keys, "NumKeys");

  std::size_t value_size_out = 0;

  thrust::device_vector<std::size_t> d_offsets(num_keys);

  hash_table.retrieve(d_prob_keys.data().get(),
                      num_keys,
                      d_offsets.data().get(),
                      d_offsets.data().get() + 1,
                      nullptr,
                      value_size_out);
  hipDeviceSynchronize();

  thrust::device_vector<Value> d_results(value_size_out);

  state.exec(nvbench::exec_tag::sync | nvbench::exec_tag::timer,
             [&](nvbench::launch& launch, auto& timer) {
               value_size_out = 0;

               timer.start();
               hash_table.retrieve(d_prob_keys.data().get(),
                                   num_keys,
                                   d_offsets.data().get(),
                                   d_offsets.data().get() + 1,
                                   d_results.data().get(),
                                   value_size_out,
                                   launch.get_stream());
               timer.stop();

               auto status = hash_table.pop_status(launch.get_stream());
               if (status.has_any_errors()) { std::cout << status << "\n"; }
             });
}

template <typename Key, typename Value, dist_type Dist, nvbench::int32_t Multiplicity>
std::enable_if_t<(sizeof(Key) != sizeof(Value)), void> nvbench_warpcore_retrieve(
  nvbench::state& state,
  nvbench::type_list<Key, Value, nvbench::enum_type<Dist>, nvbench::enum_type<Multiplicity>>)
{
  state.skip("Key should be the same type as Value.");
}

using key_type   = nvbench::type_list<nvbench::uint32_t, nvbench::uint64_t>;
using value_type = nvbench::type_list<nvbench::uint32_t, nvbench::uint64_t>;
using d_type =
  nvbench::enum_type_list<dist_type::GAUSSIAN, dist_type::GEOMETRIC, dist_type::UNIFORM>;

using multiplicity = nvbench::enum_type_list<1, 2, 4, 8, 16, 32, 64, 128, 256>;

NVBENCH_BENCH_TYPES(nvbench_warpcore_insert,
                    NVBENCH_TYPE_AXES(key_type,
                                      value_type,
                                      nvbench::enum_type_list<dist_type::UNIFORM>,
                                      multiplicity))
  .set_name("warpcore_insert_uniform_multiplicity")
  .set_type_axes_names({"Key", "Value", "Distribution", "Multiplicity"})
  .set_max_noise(3)                            // Custom noise: 3%. By default: 0.5%.
  .add_int64_axis("NumInputs", {100'000'000})  // Total number of key/value pairs: 100'000'000
  .add_float64_axis("Occupancy", {0.8});

NVBENCH_BENCH_TYPES(nvbench_warpcore_insert,
                    NVBENCH_TYPE_AXES(key_type, value_type, d_type, nvbench::enum_type_list<8>))
  .set_name("warpcore_insert_occupancy")
  .set_type_axes_names({"Key", "Value", "Distribution", "Multiplicity"})
  .set_max_noise(3)                            // Custom noise: 3%. By default: 0.5%.
  .add_int64_axis("NumInputs", {100'000'000})  // Total number of key/value pairs: 100'000'000
  .add_float64_axis("Occupancy", nvbench::range(0.1, 0.9, 0.1));

NVBENCH_BENCH_TYPES(nvbench_warpcore_count,
                    NVBENCH_TYPE_AXES(key_type,
                                      value_type,
                                      nvbench::enum_type_list<dist_type::UNIFORM>,
                                      multiplicity))
  .set_name("warpcore_count_uniform_multiplicity")
  .set_type_axes_names({"Key", "Value", "Distribution", "Multiplicity"})
  .set_timeout(100)                            // Custom timeout: 100 s. Default is 15 s.
  .set_max_noise(3)                            // Custom noise: 3%. By default: 0.5%.
  .add_int64_axis("NumInputs", {100'000'000})  // Total number of key/value pairs: 100'000'000
  .add_float64_axis("Occupancy", {0.8})
  .add_float64_axis("MatchingRate", {0.5});

NVBENCH_BENCH_TYPES(nvbench_warpcore_count,
                    NVBENCH_TYPE_AXES(key_type, value_type, d_type, nvbench::enum_type_list<8>))
  .set_name("warpcore_count_occupancy")
  .set_type_axes_names({"Key", "Value", "Distribution", "Multiplicity"})
  .set_timeout(100)                            // Custom timeout: 100 s. Default is 15 s.
  .set_max_noise(3)                            // Custom noise: 3%. By default: 0.5%.
  .add_int64_axis("NumInputs", {100'000'000})  // Total number of key/value pairs: 100'000'000
  .add_float64_axis("Occupancy", nvbench::range(0.1, 0.9, 0.1))
  .add_float64_axis("MatchingRate", {0.5});

NVBENCH_BENCH_TYPES(nvbench_warpcore_count,
                    NVBENCH_TYPE_AXES(key_type, value_type, d_type, nvbench::enum_type_list<8>))
  .set_name("warpcore_count_matching_rate")
  .set_type_axes_names({"Key", "Value", "Distribution", "Multiplicity"})
  .set_timeout(100)                            // Custom timeout: 100 s. Default is 15 s.
  .set_max_noise(3)                            // Custom noise: 3%. By default: 0.5%.
  .add_int64_axis("NumInputs", {100'000'000})  // Total number of key/value pairs: 100'000'000
  .add_float64_axis("Occupancy", {0.8})
  .add_float64_axis("MatchingRate", {0.01, 0.05, 0.1, 0.2, 0.3, 0.4, 0.5, 0.6, 0.7, 0.8, 0.9, 1});

NVBENCH_BENCH_TYPES(nvbench_warpcore_retrieve,
                    NVBENCH_TYPE_AXES(key_type,
                                      value_type,
                                      nvbench::enum_type_list<dist_type::UNIFORM>,
                                      multiplicity))
  .set_name("warpcore_retrieve_uniform_multiplicity")
  .set_type_axes_names({"Key", "Value", "Distribution", "Multiplicity"})
  .set_timeout(100)                            // Custom timeout: 100 s. Default is 15 s.
  .set_max_noise(3)                            // Custom noise: 3%. By default: 0.5%.
  .add_int64_axis("NumInputs", {100'000'000})  // Total number of key/value pairs: 100'000'000
  .add_float64_axis("Occupancy", {0.8})
  .add_float64_axis("MatchingRate", {0.5});

NVBENCH_BENCH_TYPES(nvbench_warpcore_retrieve,
                    NVBENCH_TYPE_AXES(key_type, value_type, d_type, nvbench::enum_type_list<8>))
  .set_name("warpcore_retrieve_occupancy")
  .set_type_axes_names({"Key", "Value", "Distribution", "Multiplicity"})
  .set_timeout(100)                            // Custom timeout: 100 s. Default is 15 s.
  .set_max_noise(3)                            // Custom noise: 3%. By default: 0.5%.
  .add_int64_axis("NumInputs", {100'000'000})  // Total number of key/value pairs: 100'000'000
  .add_float64_axis("Occupancy", nvbench::range(0.1, 0.9, 0.1))
  .add_float64_axis("MatchingRate", {0.5});

NVBENCH_BENCH_TYPES(nvbench_warpcore_retrieve,
                    NVBENCH_TYPE_AXES(key_type, value_type, d_type, nvbench::enum_type_list<8>))
  .set_name("warpcore_retrieve_matching_rate")
  .set_type_axes_names({"Key", "Value", "Distribution", "Multiplicity"})
  .set_timeout(100)                            // Custom timeout: 100 s. Default is 15 s.
  .set_max_noise(3)                            // Custom noise: 3%. By default: 0.5%.
  .add_int64_axis("NumInputs", {100'000'000})  // Total number of key/value pairs: 100'000'000
  .add_float64_axis("Occupancy", {0.8})
  .add_float64_axis("MatchingRate", {0.01, 0.05, 0.1, 0.2, 0.3, 0.4, 0.5, 0.6, 0.7, 0.8, 0.9, 1});
