/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <random>

#include <thrust/device_vector.h>
#include <nvbench/nvbench.cuh>

#include <cuco/static_multimap.cuh>
#include <key_generator.hpp>

NVBENCH_DECLARE_ENUM_TYPE_STRINGS(
  // Enum type:
  dist_type,
  // Callable to generate input strings:
  // Short identifier used for tables, command-line args, etc.
  // Used when context is available to figure out the enum type.
  [](dist_type d) {
    switch (d) {
      case dist_type::GAUSSIAN: return "GAUSSIAN";
      case dist_type::GEOMETRIC: return "GEOMETRIC";
      case dist_type::UNIFORM: return "UNIFORM";
      default: return "ERROR";
    }
  },
  // Callable to generate descriptions:
  // If non-empty, these are used in `--list` to describe values.
  // Used when context may not be available to figure out the type from the
  // input string.
  // Just use `[](auto) { return std::string{}; }` if you don't want these.
  [](auto) { return std::string{}; })

/**
 * @brief A benchmark evaluating multi-value `insert` performance:
 * - Total number of insertions: 100'000'000
 * - CG size: 8
 */
template <typename Key, typename Value, dist_type Dist, nvbench::int32_t Multiplicity>
std::enable_if_t<(sizeof(Key) == sizeof(Value)), void> nvbench_static_multimap_insert(
  nvbench::state& state,
  nvbench::type_list<Key, Value, nvbench::enum_type<Dist>, nvbench::enum_type<Multiplicity>>)
{
  auto const num_keys  = state.get_int64("NumInputs");
  auto const occupancy = state.get_float64("Occupancy");

  std::size_t const size = num_keys / occupancy;
  auto const cg_size     = 8;

  std::vector<Key> h_keys(num_keys);
  std::vector<cuco::pair_type<Key, Value>> h_pairs(num_keys);

  generate_keys<Dist, Multiplicity, Key>(h_keys.begin(), h_keys.end());

  for (auto i = 0; i < num_keys; ++i) {
    Key key           = h_keys[i];
    Value val         = h_keys[i];
    h_pairs[i].first  = key;
    h_pairs[i].second = val;
  }

  thrust::device_vector<cuco::pair_type<Key, Value>> d_pairs(h_pairs);

  state.add_element_count(num_keys, "NumKeys");

  state.exec(nvbench::exec_tag::sync | nvbench::exec_tag::timer,
             [&](nvbench::launch& launch, auto& timer) {
               cuco::static_multimap<Key, Value, cg_size> map{size, -1, -1};

               // Use timers to explicitly mark the target region
               timer.start();
               map.insert(d_pairs.begin(), d_pairs.end(), launch.get_stream());
               timer.stop();
             });
}

template <typename Key, typename Value, dist_type Dist, nvbench::int32_t Multiplicity>
std::enable_if_t<(sizeof(Key) != sizeof(Value)), void> nvbench_static_multimap_insert(
  nvbench::state& state,
  nvbench::type_list<Key, Value, nvbench::enum_type<Dist>, nvbench::enum_type<Multiplicity>>)
{
  state.skip("Key should be the same type as Value.");
}

/**
 * @brief A benchmark evaluating multi-value `count` performance:
 * - Total number of insertions: 100'000'000
 * - CG size: 8
 */
template <typename Key, typename Value, dist_type Dist, nvbench::int32_t Multiplicity>
std::enable_if_t<(sizeof(Key) == sizeof(Value)), void> nvbench_static_multimap_count(
  nvbench::state& state,
  nvbench::type_list<Key, Value, nvbench::enum_type<Dist>, nvbench::enum_type<Multiplicity>>)
{
  auto const num_keys      = state.get_int64("NumInputs");
  auto const occupancy     = state.get_float64("Occupancy");
  auto const matching_rate = state.get_float64("MatchingRate");

  std::size_t const size = num_keys / occupancy;
  auto const cg_size     = 8;

  std::vector<Key> h_keys(num_keys);
  std::vector<cuco::pair_type<Key, Value>> h_pairs(num_keys);

  generate_keys<Dist, Multiplicity, Key>(h_keys.begin(), h_keys.end());

  for (auto i = 0; i < num_keys; ++i) {
    Key key           = h_keys[i];
    Value val         = h_keys[i];
    h_pairs[i].first  = key;
    h_pairs[i].second = val;
  }

  generate_prob_keys<Key>(matching_rate, h_keys.begin(), h_keys.end());

  thrust::device_vector<Key> d_keys(h_keys);
  thrust::device_vector<cuco::pair_type<Key, Value>> d_pairs(h_pairs);

  state.add_element_count(num_keys, "NumKeys");

  cuco::static_multimap<Key, Value, cg_size> map{size, -1, -1};
  map.insert(d_pairs.begin(), d_pairs.end());

  state.exec(nvbench::exec_tag::sync | nvbench::exec_tag::timer,
             [&](nvbench::launch& launch, auto& timer) {
               timer.start();
               auto count = map.count(d_keys.begin(), d_keys.end(), launch.get_stream());
               timer.stop();
             });
}

template <typename Key, typename Value, dist_type Dist, nvbench::int32_t Multiplicity>
std::enable_if_t<(sizeof(Key) != sizeof(Value)), void> nvbench_static_multimap_count(
  nvbench::state& state,
  nvbench::type_list<Key, Value, nvbench::enum_type<Dist>, nvbench::enum_type<Multiplicity>>)
{
  state.skip("Key should be the same type as Value.");
}

/**
 * @brief A benchmark evaluating multi-value `find_all` performance:
 * - Total number of insertions: 100'000'000
 * - CG size: 8
 */
template <typename Key, typename Value, dist_type Dist, nvbench::int32_t Multiplicity>
std::enable_if_t<(sizeof(Key) == sizeof(Value)), void> nvbench_static_multimap_find_all(
  nvbench::state& state,
  nvbench::type_list<Key, Value, nvbench::enum_type<Dist>, nvbench::enum_type<Multiplicity>>)
{
  auto const num_keys      = state.get_int64("NumInputs");
  auto const occupancy     = state.get_float64("Occupancy");
  auto const matching_rate = state.get_float64("MatchingRate");

  std::size_t const size = num_keys / occupancy;
  auto const cg_size     = 8;

  std::vector<Key> h_keys(num_keys);
  std::vector<cuco::pair_type<Key, Value>> h_pairs(num_keys);

  generate_keys<Dist, Multiplicity, Key>(h_keys.begin(), h_keys.end());

  for (auto i = 0; i < num_keys; ++i) {
    Key key           = h_keys[i];
    Value val         = h_keys[i];
    h_pairs[i].first  = key;
    h_pairs[i].second = val;
  }

  generate_prob_keys<Key>(matching_rate, h_keys.begin(), h_keys.end());

  thrust::device_vector<Key> d_keys(h_keys);
  thrust::device_vector<cuco::pair_type<Key, Value>> d_pairs(h_pairs);

  state.add_element_count(num_keys, "NumKeys");

  cuco::static_multimap<Key, Value, cg_size> map{size, -1, -1};
  map.insert(d_pairs.begin(), d_pairs.end());

  auto num_matches        = map.count(d_keys.begin(), d_keys.end());
  std::size_t output_size = num_matches + num_keys;
  thrust::device_vector<cuco::pair_type<Key, Value>> d_results(output_size);

  state.exec(
    nvbench::exec_tag::sync | nvbench::exec_tag::timer, [&](nvbench::launch& launch, auto& timer) {
      timer.start();
      map.find_all(d_keys.begin(), d_keys.end(), d_results.data().get(), launch.get_stream());
      timer.stop();
    });
}

template <typename Key, typename Value, dist_type Dist, nvbench::int32_t Multiplicity>
std::enable_if_t<(sizeof(Key) != sizeof(Value)), void> nvbench_static_multimap_find_all(
  nvbench::state& state,
  nvbench::type_list<Key, Value, nvbench::enum_type<Dist>, nvbench::enum_type<Multiplicity>>)
{
  state.skip("Key should be the same type as Value.");
}

template <typename Key, typename Value, dist_type Dist, nvbench::int32_t Multiplicity>
std::enable_if_t<(sizeof(Key) == sizeof(Value)), void> nvbench_static_multimap_retrieve(
  nvbench::state& state,
  nvbench::type_list<Key, Value, nvbench::enum_type<Dist>, nvbench::enum_type<Multiplicity>>)
{
  auto const num_keys      = state.get_int64("NumInputs");
  auto const occupancy     = state.get_float64("Occupancy");
  auto const matching_rate = state.get_float64("MatchingRate");

  std::size_t const size = num_keys / occupancy;
  auto const cg_size     = 8;

  std::vector<Key> h_keys(num_keys);
  std::vector<cuco::pair_type<Key, Value>> h_pairs(num_keys);

  generate_keys<Dist, Multiplicity, Key>(h_keys.begin(), h_keys.end());

  for (auto i = 0; i < num_keys; ++i) {
    Key key           = h_keys[i];
    Value val         = h_keys[i];
    h_pairs[i].first  = key;
    h_pairs[i].second = val;
  }

  generate_prob_keys<Key>(matching_rate, h_keys.begin(), h_keys.end());

  thrust::device_vector<Key> d_keys(h_keys);
  thrust::device_vector<cuco::pair_type<Key, Value>> d_pairs(h_pairs);

  state.add_element_count(num_keys, "NumKeys");

  cuco::static_multimap<Key, Value, cg_size> map{size, -1, -1};
  map.insert(d_pairs.begin(), d_pairs.end());

  auto num_matches        = map.count(d_keys.begin(), d_keys.end());
  std::size_t output_size = num_matches + num_keys;
  thrust::device_vector<cuco::pair_type<Key, Value>> d_results(output_size);

  state.exec(
    nvbench::exec_tag::sync | nvbench::exec_tag::timer, [&](nvbench::launch& launch, auto& timer) {
      timer.start();
      auto count = map.count(d_keys.begin(), d_keys.end(), launch.get_stream());
      map.find_all(d_keys.begin(), d_keys.end(), d_results.data().get(), launch.get_stream());
      timer.stop();
    });
}

template <typename Key, typename Value, dist_type Dist, nvbench::int32_t Multiplicity>
std::enable_if_t<(sizeof(Key) != sizeof(Value)), void> nvbench_static_multimap_retrieve(
  nvbench::state& state,
  nvbench::type_list<Key, Value, nvbench::enum_type<Dist>, nvbench::enum_type<Multiplicity>>)
{
  state.skip("Key should be the same type as Value.");
}

/**
 * @brief A benchmark evaluating multi-value retrieve (`count` + `find_all`) performance:
 * - Total number of insertions: 100'000'000
 * - CG size: 8
 */

using key_type   = nvbench::type_list<nvbench::int32_t>;
using value_type = nvbench::type_list<nvbench::int32_t>;
using d_type =
  nvbench::enum_type_list<dist_type::GAUSSIAN, dist_type::GEOMETRIC, dist_type::UNIFORM>;

using multiplicity = nvbench::enum_type_list<1, 64>;

NVBENCH_BENCH_TYPES(nvbench_static_multimap_insert,
                    NVBENCH_TYPE_AXES(key_type,
                                      value_type,
                                      nvbench::enum_type_list<dist_type::UNIFORM>,
                                      multiplicity))
  .set_name("staic_multimap_insert_uniform_multiplicity")
  .set_type_axes_names({"Key", "Value", "Distribution", "Multiplicity"})
  .set_max_noise(3)                            // Custom noise: 3%. By default: 0.5%.
  .add_int64_axis("NumInputs", {100'000'000})  // Total number of key/value pairs: 100'000'000
  .add_float64_axis("Occupancy", {0.8});

NVBENCH_BENCH_TYPES(nvbench_static_multimap_count,
                    NVBENCH_TYPE_AXES(key_type,
                                      value_type,
                                      nvbench::enum_type_list<dist_type::UNIFORM>,
                                      multiplicity))
  .set_name("staic_multimap_count_uniform_multiplicity")
  .set_type_axes_names({"Key", "Value", "Distribution", "Multiplicity"})
  .set_timeout(100)                            // Custom timeout: 100 s. Default is 15 s.
  .set_max_noise(3)                            // Custom noise: 3%. By default: 0.5%.
  .add_int64_axis("NumInputs", {100'000'000})  // Total number of key/value pairs: 100'000'000
  .add_float64_axis("Occupancy", {0.8})
  .add_float64_axis("MatchingRate", {0.5});

NVBENCH_BENCH_TYPES(nvbench_static_multimap_find_all,
                    NVBENCH_TYPE_AXES(key_type,
                                      value_type,
                                      nvbench::enum_type_list<dist_type::UNIFORM>,
                                      multiplicity))
  .set_name("staic_multimap_find_all_uniform_multiplicity")
  .set_type_axes_names({"Key", "Value", "Distribution", "Multiplicity"})
  .set_timeout(100)                            // Custom timeout: 100 s. Default is 15 s.
  .set_max_noise(3)                            // Custom noise: 3%. By default: 0.5%.
  .add_int64_axis("NumInputs", {100'000'000})  // Total number of key/value pairs: 100'000'000
  .add_float64_axis("Occupancy", {0.8})
  .add_float64_axis("MatchingRate", {0.5});
